#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <algorithm>
#include <stdlib.h>

//#define N 4
#define BLOCK_SIZE 4
#define GRID_SIZE 2


using namespace std;

__device__ volatile int g_mutex;

void cuda_error_check(hipError_t err , const char *msg )
{
    if(err != hipSuccess)
    {
      printf("The error is %s, %s \n", hipGetErrorString(err), msg );
      exit(1);
    }
}

__device__ void __gpu_sync(int goalVal)
{
    //thread ID in a block
    int tid_in_block = threadIdx.x * blockDim.y + threadIdx.y;
    // only thread 0 is used for synchronization
    if (tid_in_block == 0)
    {
            atomicAdd((int *)&g_mutex, 1);

            while(g_mutex != goalVal)
            {
            //Do nothing here
            }        
    }
    
    //__threadfence();
    __syncthreads();
}

__global__ void matrix(int *d_a)
{

    int row = (blockIdx.y * blockDim.y) + threadIdx.y ; 
    int col = (blockIdx.x * blockDim.x) + threadIdx.x ;
    //int L1  = blockDim.x ;
    //int L2  = blockDim.y ;
      
    int offset = col + row * blockDim.x * gridDim.x ;

    int blockid = blockIdx.y * gridDim.x + blockIdx.x ;
    int off = col + row * gridDim.x  - blockDim.x*blockid ;

    off += (blockDim.x - gridDim.x ) * threadIdx.y;

    int off_x = off - (threadIdx.y * blockDim.x) ;
    int off_y = (off - threadIdx.x) / blockDim.x ;

    //while(blockid!=g_mutex ){}

    if(blockid==0)
            d_a[offset] = g_mutex ;  


    __gpu_sync(4);

    if(blockid!=0)
            d_a[offset] = g_mutex ;  
/*
    if(blockid == g_mutex && off==0)
    {   atomicAdd((int *)&g_mutex, 1);
    }*/

}

int main(int argc , char **argv)
{

    int a[(GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE];
    int i,j;
    //int m,n;
     
    for(j=0;j<(GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE;j++)
    {
        a[j]=0;
    }

    int *d_a ;

    cuda_error_check(hipSetDevice(0) , "hipSetDevice failed!" );

    cuda_error_check(hipMalloc((void **)&d_a , (GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE* sizeof(int)),"hipMalloc Failed!");
    cuda_error_check(hipMemcpy(d_a , a , (GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE * sizeof(int) , hipMemcpyHostToDevice),"hipMemcpy H-D failed!");

    dim3 dimBlock(BLOCK_SIZE , BLOCK_SIZE );
    dim3 dimGrid(GRID_SIZE , GRID_SIZE);

    matrix<<< dimGrid , dimBlock >>>(d_a);

    cuda_error_check(hipMemcpy(a , d_a , (GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE * sizeof(int) , hipMemcpyDeviceToHost),"hipMemcpy D-H failed!");


for(j=0;j<(GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE;j++)
{
    if(j%(BLOCK_SIZE*GRID_SIZE) == 0)
        cout<<endl;
    if(j%(GRID_SIZE*BLOCK_SIZE*BLOCK_SIZE)==0)
        cout<<endl;
    if(j%BLOCK_SIZE ==0)
        cout<<"  ";
        
    cout.width( 3 );
    cout<<a[j]<<" ";
}
            printf("\n\n");

 /*for(m=0;m<2;m++)
 {
    for(n=0;n<2;n++)
    {*/
   /*
         for(i=0;i<2*BLOCK_SIZE;i++)
         {
            for(j=0;j<2*BLOCK_SIZE;j++)
            {                  // [m*2+n]
                //printf("%d\t",a[i*BLOCK_SIZE + j]);
                cout.width( 3 );
                //cout<<a[(m*2+n)*16 + i*BLOCK_SIZE + j]<<" ";
                cout<<a[i*(2*BLOCK_SIZE) + j]<<" ";

            }
            printf("\n");
         }

         */
        /*    printf("\n");
    }
}
*/
return 0;
}





 /*      
if(blockIdx.x == 0 && blockIdx.y == 1)
    d_a[offset] = offset;
else
    d_a[offset] = -1;

if(row==0)
        d_a[offset] = -1 * offset;
if(col==0)
        d_a[offset] = -1 * offset/(BLOCK_SIZE*BLOCK_SIZE);
*/