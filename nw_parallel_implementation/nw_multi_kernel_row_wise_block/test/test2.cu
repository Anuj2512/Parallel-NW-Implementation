#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <algorithm>
#include <stdlib.h>

#define N 4
#define BLOCK_SIZE 4
#define GRID_SIZE 1

void cuda_error_check(hipError_t err , const char *msg )
{
    if(err != hipSuccess)
    {
      printf("The error is %s, %s \n", hipGetErrorString(err), msg );
      exit(1);
    }
}

__global__ void matrix(int *d_a)
{

int row = (blockIdx.y * blockDim.y) + threadIdx.y ; 
int col = (blockIdx.x * blockDim.x) + threadIdx.x ;
int L1  = blockDim.x ;
int L2  = blockDim.y ;
       
__shared__ int s_a[N*N];


s_a[row*L1 + col] =  d_a[row*L1 + col] ;

__syncthreads();


d_a[row*L1 + col] = s_a[row*L1 + col] ;

/*if(row == 0)
{
    d_a[col] = col * -1 ;
}else if(col == 0)
{
    d_a[row*L1] = row * -1 ;
}else
{
    d_a[row*L1 + col] = 0 ;
}

__syncthreads();

        int z ;
        for( z = 2; z <= L1 + L2 - 1; z++ )  
        {
            
            int mx = ( 1>z-L1 ? 1  : z-L1);
            int mn = ( L1<z-1 ? L1 : z-1 );
            if(z>L1)    mx++ ;


                if(row >=mx && row<=mn)
                {
                    d_a[row*L1 + (z-row)] = d_a[(row-1)*L1 + (z-row)] + d_a[row*L1 + (z-row-1)]   ;
                }

        __syncthreads();
        
        }
*/

}

int main(int argc , char **argv)
{

int a[N*N];
int i,j;


 


int *d_a ;


    cuda_error_check(hipSetDevice(0) , "hipSetDevice failed!" );

    cuda_error_check(hipMalloc((void **)&d_a , N*N* sizeof(int)),"hipMalloc Failed!");
    cuda_error_check(hipMemcpy(d_a , a , N*N * sizeof(int) , hipMemcpyHostToDevice),"hipMemcpy H-D failed!");

    dim3 dimBlock(BLOCK_SIZE , BLOCK_SIZE );
    dim3 dimGrid(GRID_SIZE , GRID_SIZE);

    matrix<<< dimGrid , dimBlock >>>(d_a);

    cuda_error_check(hipMemcpy(a , d_a , N*N * sizeof(int) , hipMemcpyDeviceToHost),"hipMemcpy D-H failed!");
 
printf("\n\n");

 for(i=0;i<N;i++)
 {
    for(j=0;j<N;j++)
    {
        //printf("%d\t",a[i*N + j]);
        printf("%d\t",a[i*N + j]);
    }
    printf("\n");
 }
return 0;
}
