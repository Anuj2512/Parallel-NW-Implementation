#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <algorithm>
#include <stdlib.h>

//#define N 4
#define BLOCK_SIZE 4
#define GRID_SIZE 2


// threadfence();

using namespace std;

__device__ volatile int Arrayin[100];
__device__ volatile int Arrayout[100];


void cuda_error_check(hipError_t err , const char *msg )
{
    if(err != hipSuccess)
    {
      printf("The error is %s, %s \n", hipGetErrorString(err), msg );
      exit(1);
    }
}

__device__ void __gpu_sync(int goalVal , volatile int *Arrayin , volatile int *Arrayout)
{
    // thread ID in a block
    int tid_in_blk = threadIdx.x * blockDim.y  + threadIdx.y;
    int nBlockNum = gridDim.x * gridDim.y;
    int bid = blockIdx.x * gridDim.y + blockIdx.y;


    // only thread 0 is used for synchronization
    if (tid_in_blk == 0)
    {
        Arrayin[bid] = goalVal;
    }
    
    if (bid == 1)
    {
        if (tid_in_blk < nBlockNum)
        {
            while (Arrayin[tid_in_blk] != goalVal){
            //Do nothing here
            }
        }

        __syncthreads();
        __threadfence();

        if (tid_in_blk < nBlockNum)
        {
            Arrayout[tid_in_blk] = goalVal;
        }
    }
    if (tid_in_blk == 0)
    {
            while (Arrayout[bid] != goalVal){
            //Do nothing here
            }
    }

    __syncthreads();
    __threadfence();

}

__global__ void matrix(int *d_a)
{

    int row = (blockIdx.y * blockDim.y) + threadIdx.y ; 
    int col = (blockIdx.x * blockDim.x) + threadIdx.x ;
    //int L1  = blockDim.x ;
    //int L2  = blockDim.y ;
      
    int offset = col + row * blockDim.x * gridDim.x ;

    int blockid = blockIdx.y * gridDim.x + blockIdx.x ;
    int off = col + row * gridDim.x  - blockDim.x*blockid ;

    off += (blockDim.x - gridDim.x ) * threadIdx.y;

    int off_x = off - (threadIdx.y * blockDim.x) ;
    int off_y = (off - threadIdx.x) / blockDim.x ;



    //while(blockid!=g_mutex ){}
    __gpu_sync(4, Arrayin , Arrayout) ;

    d_a[offset] = 0 ;  

    /*if(blockid == g_mutex && off==0)
    {   atomicAdd((int *)&g_mutex, 1);
    }*/

}

int main(int argc , char **argv)
{

    int a[(GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE];
    int i,j;
    //int m,n;
     
    for(j=0;j<(GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE;j++)
    {
        a[j]=0;
    }

    int *d_a ;

    cuda_error_check(hipSetDevice(0) , "hipSetDevice failed!" );

    hipDeviceReset();

    cuda_error_check(hipMalloc((void **)&d_a , (GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE* sizeof(int)),"hipMalloc Failed!");
    cuda_error_check(hipMemcpy(d_a , a , (GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE * sizeof(int) , hipMemcpyHostToDevice),"hipMemcpy H-D failed!");

    dim3 dimBlock(BLOCK_SIZE , BLOCK_SIZE );
    dim3 dimGrid(GRID_SIZE , GRID_SIZE);

    matrix<<< dimGrid , dimBlock >>>(d_a);

    cuda_error_check(hipMemcpy(a , d_a , (GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE * sizeof(int) , hipMemcpyDeviceToHost),"hipMemcpy D-H failed!");


for(j=0;j<(GRID_SIZE*GRID_SIZE)*BLOCK_SIZE*BLOCK_SIZE;j++)
{
    if(j%(BLOCK_SIZE*GRID_SIZE) == 0)
        cout<<endl;
    if(j%(GRID_SIZE*BLOCK_SIZE*BLOCK_SIZE)==0)
        cout<<endl;
    if(j%BLOCK_SIZE ==0)
        cout<<"  ";
        
    cout.width( 3 );
    cout<<a[j]<<" ";
}
            printf("\n\n");

 /*for(m=0;m<2;m++)
 {
    for(n=0;n<2;n++)
    {*/
   /*
         for(i=0;i<2*BLOCK_SIZE;i++)
         {
            for(j=0;j<2*BLOCK_SIZE;j++)
            {                  // [m*2+n]
                //printf("%d\t",a[i*BLOCK_SIZE + j]);
                cout.width( 3 );
                //cout<<a[(m*2+n)*16 + i*BLOCK_SIZE + j]<<" ";
                cout<<a[i*(2*BLOCK_SIZE) + j]<<" ";

            }
            printf("\n");
         }

         */
        /*    printf("\n");
    }
}
*/
return 0;
}

