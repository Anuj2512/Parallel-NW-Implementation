#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <algorithm>
#include <stdlib.h>

#define N         40
#define GAP       -2
#define MATCH      1
#define MISMATCH  -1

//#include "kernels.h"

using namespace std ;


__device__ volatile int g_mutex;
__device__ volatile int g_mutex_sync;

__device__ void __gpu_sync(int goalVal)
{
    //thread ID in a block
    int tid_in_block = threadIdx.x * blockDim.y + threadIdx.y;
    // only thread 0 is used for synchronization
    if (tid_in_block == 0)
    {
            atomicAdd((int *)&g_mutex_sync, 1);

            while(g_mutex_sync != goalVal)
            {
            //Do nothing here
            }        
            //g_mutex = 0 ;
    }
    
    //__threadfence();
    __syncthreads();

}

__global__ void calc_matrix(int *d_mat , int l1 , int l2)
{  
    int row = (blockIdx.y * blockDim.y) + threadIdx.y ; 

    int col = (blockIdx.x * blockDim.x) + threadIdx.x ;

    int offset = col + row * blockDim.x * gridDim.x ;

    __gpu_sync(1);

    d_mat[offset]       =  gridDim.x     ;

}



void cuda_error_check(hipError_t err , const char *msg );

int main()
{
        const int NN = 64 ;
        const int  L1 = NN ;
        const int  L2 = NN ;
           

      //  int  mat[L1*L2];        // Dynamic Prog. Matrix

        int *mat = new int[L1*L2] ;
        

        //printf("\nSize : %d\n",sizeof(mat));
// #################################  Parallel #####################################################        
        int *d_mat ;
        
        cuda_error_check(hipSetDevice(0) , "hipSetDevice failed!" );


        cuda_error_check(hipMalloc((void **)&d_mat       , L1 * L2 * sizeof(int)),"hipMalloc Failed!");

        int BLOCK_SIZE = 16;
        int GRID_SIZE  = 1  ;

        dim3 dimBlock(BLOCK_SIZE , BLOCK_SIZE );
        dim3 dimGrid(GRID_SIZE , 1);

        calc_matrix<<< dimBlock , dimGrid >>> ( d_mat , L1 , L2 ) ;

        printf("\nKernel Complete Size : %d %d : %d ", L1 ,L2 , L1*L2 );

        cuda_error_check(hipMemcpy(mat , d_mat ,L1* L2 * sizeof(int) , hipMemcpyDeviceToHost),"hipMemcpy D-H failed! 1");
        printf("\nCudamemcpy D-H Complete");

        hipFree(d_mat);

       // cudaDeviceReset();

// #################################################################################################        

        printf("\nMatrix: \n");

        cout << "        ";
        for( int j = 0; j < L1; j++ )
        {
                //cout << seq_2[ j ] << "   ";
        }
        cout << "\n";

        for( int i = 0; i < L2; i++ )
        {
                if( i > 0 )
                {
                        //cout << seq_1[ i-1 ] << " ";
                }
                for( int j = 0; j < L1; j++ )
                {
                        cout.width(3);
                        cout << mat[i*L1 + j] << " ";
                }
                cout << endl;
        }

// #################################################################################################
        
}

void cuda_error_check(hipError_t err , const char *msg )
{
    if(err != hipSuccess)
    {
      printf("The error is %s, %s \n", hipGetErrorString(err), msg );
      exit(1);
    }
}
